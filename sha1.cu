/*** 
 * @Author       : FeiYehua
 * @Date         : 2025-01-01 12:47:59
 * @LastEditTime : 2025-01-01 12:48:02
 * @LastEditors  : FeiYehua
 * @Description  : 
 * @FilePath     : sha1.cpp
 * @     © 2024 FeiYehua
 */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>
#include <cstring>
using namespace std;
extern __constant__ uint32_t target[5];
// 32-bit rotate
__device__ inline uint32_t ROT(uint32_t x, int n) {
    return ((x << n) | (x >> (32 - n)));
}

// SHA init constants
#define I1 1732584193U
#define I2 4023233417U
#define I3 2562383102U
#define I4 271733878U
#define I5 3285377520U

// Main loop SHA logical functions f1 to f4
__device__ inline uint32_t f1(uint32_t x, uint32_t y, uint32_t z)
{
    return ((x & y) | (~x & z));
}
__device__ inline uint32_t f2(uint32_t x, uint32_t y, uint32_t z) { return (x ^ y ^ z); }
__device__ inline uint32_t f3(uint32_t x, uint32_t y, uint32_t z)
{
    return ((x & y) | (x & z) | (y & z));
}
__device__ inline uint32_t f4(uint32_t x, uint32_t y, uint32_t z) { return (x ^ y ^ z); }

// Calculation functions for 80 rounds of SHA1
#define CALC1(i)                                                               \
    temp = ROT(A, 5) + f1(B, C, D) + mes[i] + E + 1518500249U;                     \
    E = D;                                                                       \
    D = C;                                                                       \
    C = ROT(B, 30);                                                              \
    B = A;                                                                       \
    A = temp

#define CALC2(i)                                                               \
    temp = ROT(A, 5) + f2(B, C, D) + mes[i] + E + 1859775393U;                     \
    E = D;                                                                       \
    D = C;                                                                       \
    C = ROT(B, 30);                                                              \
    B = A;                                                                       \
    A = temp

#define CALC3(i)                                                               \
    temp = ROT(A, 5) + f3(B, C, D) + mes[i] + E + 2400959708U;                     \
    E = D;                                                                       \
    D = C;                                                                       \
    C = ROT(B, 30);                                                              \
    B = A;                                                                       \
    A = temp

#define CALC4(i)                                                               \
    temp = ROT(A, 5) + f4(B, C, D) + mes[i] + E + 3395469782U;                     \
    E = D;                                                                       \
    D = C;                                                                       \
    C = ROT(B, 30);                                                              \
    B = A;                                                                       \
    A = temp



// 我们考虑计算一个16位16进制串的sha1.
// 也就是消息是一个64位整数。
// 需要计算的消息长度是64bit。
// 我们每次分配一个block计算2^16个sha1，
// 每个grid总计算2^16*2^16个sha1，
// 那么每个block的前48bit是可以确定的

// 初始化消息，附加填充位
//mes1是前32bit，mes2是后32bit
__device__ uint64_t getSha1(uint32_t mes1,uint32_t mes2)
{
    uint32_t mes[80];
    // mes = (uint32_t *)malloc(sizeof(uint32_t) * 64);
    // cudaMalloc(&mes,sizeof(uint32_t)*64);

    mes[0] = mes1;
    mes[1] = mes2;
    mes[2] = 1U << 31;
#pragma unroll
    for (int i = 3; i <= 14; i++)
    {
        mes[i] = 0;
    }
    mes[15] = 64;

#pragma unroll
    for (int i = 16; i < 80; i++)
    {
        mes[i] = ROT((mes[i - 3] ^ mes[i - 8] ^ mes[i - 14] ^ mes[i - 16]), 1);
    }
    uint32_t A,B,C,D,E,temp;
        // Perform sha calculation
    A = I1;
    B = I2;
    C = I3;
    D = I4;
    E = I5;

    // 80 rounds
    CALC1(0);
    CALC1(1);
    CALC1(2);
    CALC1(3);
    CALC1(4);
    CALC1(5);
    CALC1(6);
    CALC1(7);
    CALC1(8);
    CALC1(9);
    CALC1(10);
    CALC1(11);
    CALC1(12);
    CALC1(13);
    CALC1(14);
    CALC1(15);
    CALC1(16);
    CALC1(17);
    CALC1(18);
    CALC1(19);
    CALC2(20);
    CALC2(21);
    CALC2(22);
    CALC2(23);
    CALC2(24);
    CALC2(25);
    CALC2(26);
    CALC2(27);
    CALC2(28);
    CALC2(29);
    CALC2(30);
    CALC2(31);
    CALC2(32);
    CALC2(33);
    CALC2(34);
    CALC2(35);
    CALC2(36);
    CALC2(37);
    CALC2(38);
    CALC2(39);
    CALC3(40);
    CALC3(41);
    CALC3(42);
    CALC3(43);
    CALC3(44);
    CALC3(45);
    CALC3(46);
    CALC3(47);
    CALC3(48);
    CALC3(49);
    CALC3(50);
    CALC3(51);
    CALC3(52);
    CALC3(53);
    CALC3(54);
    CALC3(55);
    CALC3(56);
    CALC3(57);
    CALC3(58);
    CALC3(59);
    CALC4(60);
    CALC4(61);
    CALC4(62);
    CALC4(63);
    CALC4(64);
    CALC4(65);
    CALC4(66);
    CALC4(67);
    CALC4(68);
    CALC4(69);
    CALC4(70);
    CALC4(71);
    CALC4(72);
    CALC4(73);
    CALC4(74);
    CALC4(75);
    CALC4(76);
    CALC4(77);
    CALC4(78);
    CALC4(79);

    A += I1;
    B += I2;
    C += I3;
    D += I4;
    E += I5;

    // free(mes);
    // cudaFree(mes);
    // printf("%08x%08x %08x%08x%08x%08x%08x\n",mes1,mes2,A,B,C,D,E);
    // for(int i=0;i<5;i++)
    // {
    //     printf("%08x",target[i]);
    // }
    // printf("\n");
    return (A==target[0]&&B==target[1]&&C==target[2]&&D==target[3]&&E==target[4]);
    
}

//每个block的计算函数：确定消息的前48bit，计算最后16bit对应的hash
__global__ void blockSha1(uint32_t mes1,uint64_t* result)
{
    uint32_t mes2=threadIdx.x+blockIdx.x*blockDim.x;
    //printf("%d\n",mes2);
    for(int i=0;i<(1<<16);i++)
    {
        if(getSha1(mes1,(mes2<<16)+i)!=0)
        {
            *result = ((uint64_t)mes1 << 32) + (mes2<<16) + i;
            break;
        }
    }
}

#ifdef PROFILE
cudaEvent_t startTime,endTime;
float elapseTime;
#endif
//start是给定的起始点：前16bit
void cal(uint32_t start,uint64_t* result,uint32_t* calcBuffer)
{
#ifdef PROFILE
    cudaEventCreate(&startTime);
    cudaEventCreate(&endTime);
    cudaEventRecord(startTime, 0);
#endif
    blockSha1<<<(1<<8),(1<<8)>>>(start,result);
#ifdef PROFILE
    cudaEventRecord(endTime, 0);
#endif
#ifndef PROFILE
    hipDeviceSynchronize();
#endif
#ifdef PROFILE
    cudaEventSynchronize(endTime);
    cudaEventElapsedTime(&elapseTime, startTime, endTime);
    printf("%f ms\n", elapseTime);
#endif
    auto err=hipGetLastError();
    if(err!=hipSuccess)
    {
        printf("%s\n",hipGetErrorString(err));
        exit(-1);
    }
}
