/*** 
 * @Author       : FeiYehua
 * @Date         : 2025-01-01 12:52:46
 * @LastEditTime : 2025-01-01 12:52:46
 * @LastEditors  : FeiYehua
 * @Description  : 
 * @FilePath     : main.cpp
 * @     © 2024 FeiYehua
 */
#include "timer.hpp"
#include "sha1.hpp"
using namespace std;
uint32_t *target;
uint32_t hostTarget[5]={0xcb473678,0x976f425d,0x6ec13398,0x38f11011,0x007ad27d};
uint64_t *result;
uint64_t start;
uint64_t hostResult;
int main()
{
    auto fp=fopen("CurrentProgress.txt","w+");
    if(fscanf(fp,"%lu",&start)!=1)
    {
        start=0;
    }
    hipSetDevice(0);
    hipMalloc(&target,sizeof(uint32_t)*5);
    hipMalloc(&result,sizeof(uint64_t));
    hipMemcpy(target,hostTarget,sizeof(uint32_t)*5,hipMemcpyHostToDevice);
    for(uint64_t i=start;i<=((uint64_t)1<<32ll);i++)
    {
        cal(i,target,result);
        // hipMemcpy(&hostResult,result,sizeof(uint64_t),hipMemcpyDeviceToHost);
        // if(hostResult!=0)
        // {
        //     fprintf(fp,"%lu\n",hostResult);
        //     return 0;
        // }
        fseek(fp, 0L, SEEK_SET);
        fprintf(fp,"%lu\n",i);
    }
    return 0;
}