/*** 
 * @Author       : FeiYehua
 * @Date         : 2025-01-01 12:52:46
 * @LastEditTime : 2025-01-01 12:52:46
 * @LastEditors  : FeiYehua
 * @Description  : 
 * @FilePath     : main.cpp
 * @     © 2024 FeiYehua
 */
#include "timer.hpp"
#include "sha1.hpp"
using namespace std;
// uint32_t *target;
__constant__ uint32_t target[5]={0xabc21d3f,0x9d5d98ec,0xae9f3d51,0x44752901,0x71f2bbed};
uint64_t *result;
uint64_t start;
uint64_t hostResult;
uint32_t* calcBuffer;
int main()
{
    auto fp=fopen("CurrentProgress.txt","r+");
    if(fp==NULL) exit(-1);
    if(fscanf(fp,"%lu",&start)!=1)
    {
        start=0;
    }
    hipSetDevice(0);
    // hipMalloc(&target,sizeof(uint32_t)*5);
    hipMalloc(&result,sizeof(uint64_t));
    hipMalloc(&calcBuffer,sizeof(uint32_t)*80*(1<<16));
    // hipMemcpy(target,hostTarget,sizeof(uint32_t)*5,hipMemcpyHostToDevice);
    for(uint64_t i=start;i<=((uint64_t)1<<32ll);i++)
    {
        cal(i,result,calcBuffer);
        hipMemcpy(&hostResult,result,sizeof(uint64_t),hipMemcpyDeviceToHost);
        if(hostResult!=0)
        {
            auto fpResult=fopen("Result.txt","w");
            fprintf(fpResult,"%lu\n",hostResult);
            return 0;
        }
        fseek(fp, 0L, SEEK_SET);
        fprintf(fp,"%lu\n",i);
    }
    return 0;
}